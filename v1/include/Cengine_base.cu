#include "Cengine_base.hpp"

__device__ __constant__ unsigned char cg_cmem[CG_CONST_MEM_SIZE];

#ifdef _WITH_CUBLAS
#include <hipblas.h>
hipblasHandle_t Cengine_cublas;
//hipblasCreate(&Cengine_cublas);
#endif 
